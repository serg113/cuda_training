/*

to compile this code 
  >> nvcc simple_cuda_test.cu -o cuda_test
to profile execution speed
  >> nvprof cuda_test.exe
  
  runs in 456.87 us on my laptop with GeForce GT 745M

*/



#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

// this function implements grid-sride loop
__global__
void add(int n, float *x, float *y)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
	
	int stride = blockDim.x * gridDim.x;

  for (int i = index; i < n; i += stride) 
  {
		y[i] = x[i] + y[i];
	}
}

int main(void)
{
  int N = 1<<20; 
  float *x, *y;

  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  int blockSize = 256;
  int numBlocks = (N + blockSize - 1) / blockSize;

  // 4069x256 
  add<<<numBlocks, blockSize>>>(N, x, y);

  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
  {
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  }
  std::cout << "Max error: " << maxError << std::endl;

  hipFree(x);
  hipFree(y);
  
  return 0;
}
